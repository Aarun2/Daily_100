#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"

__global__ void kernel (void) {
}

__global__ void adder(int a, int b, int *c) {
    *c = a + b;
}

int main ( void ) {
    kernel<<<1,1>>>();
    printf("Hello World!\n");
    
    int c;
    int *dev_c;
    
    HANDLE_ERROR(hipMalloc((void**) &dev_c, sizeof(int) ));
    
    adder<<<1,1>>>(2, 7, dev_c);
    
    HANDLE_ERROR(hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost ));
    
    printf("2 + 7 = %d\n", c);
    
    hipFree(dev_c);
    
    return 0;
}